#include "grouped_gemm.h"

#include <ATen/cuda/HIPContext.h>
#include <c10/util/BFloat16.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

#include "cutlass/bfloat16.h"
#include "cutlass/complex.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"

#include <type_traits>

namespace grouped_gemm {

#define CUDA_CALL(code)					    \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    TORCH_CHECK(status == hipSuccess, err);		    \
  } while (0)

#define CUBLAS_CALL(code)					  \
  do {								  \
    hipblasStatus_t status = code;				  \
    TORCH_CHECK(status == HIPBLAS_STATUS_SUCCESS, "CuBLAS Error"); \
  } while (0)

#define GROUPED_GEMM_STRINGIFY_HELPER(x) #x
#define GROUPED_GEMM_STRINGIFY(x) \
  GROUPED_GEMM_STRINGIFY_HELPER(x)

template <bool trans>
using GroupedGemmInputLayout = std::conditional_t<trans, ::cutlass::layout::ColumnMajor, ::cutlass::layout::RowMajor>;

using GroupedGemmConfig = ::cutlass::gemm::device::DefaultGemmConfiguration<
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  ::cutlass::bfloat16_t,
  ::cutlass::bfloat16_t,
  ::cutlass::bfloat16_t,
  float
>;

// TODO(tgale): Update this for SM90 when it's supported by CUTLASS.
template <bool trans_a, bool trans_b>
using GroupedGemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
  // A operand.
  ::cutlass::bfloat16_t,
  GroupedGemmInputLayout<trans_a>,
  ::cutlass::ComplexTransform::kNone,
  GroupedGemmConfig::kAlignmentA,
  // B operand.
  ::cutlass::bfloat16_t,
  GroupedGemmInputLayout<trans_b>,
  ::cutlass::ComplexTransform::kNone,
  GroupedGemmConfig::kAlignmentB,
  // C operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  float,
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  GroupedGemmConfig::ThreadblockShape,
  GroupedGemmConfig::WarpShape,
  GroupedGemmConfig::InstructionShape,
  GroupedGemmConfig::EpilogueOutputOp,
  // NOTE: Threadblock swizzling is currently not supported by CUTLASS's grouped kernels.
  // This parameter is passed in at present to match the APIs of other kernels. The parameter
  // is unused within the kernel.
  ::cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
  // TODO(tgale): Experiment with GroupScheduleMode.
  // TODO(tgale): Tune this for SM90.
  GroupedGemmConfig::kStages>::GemmKernel;

template <bool trans_a, bool trans_b>
using GemmGrouped = ::cutlass::gemm::device::GemmGrouped<GroupedGemmKernel<trans_a, trans_b>>;

template <bool trans_a, bool trans_b>
std::vector<cutlass::gemm::GemmCoord> MakeProblemSizes(torch::Tensor a, torch::Tensor b, torch::Tensor batch_sizes) {
  const size_t num_experts = batch_sizes.size(0);
  const size_t hidden_in = a.size(1), hidden_out = (trans_a || trans_b) ? b.size(1) : b.size(2);
  std::vector<cutlass::gemm::GemmCoord> problem_sizes(num_experts);
  for (int i = 0; i < num_experts; ++i) {
    int64_t bs = batch_sizes.data_ptr<int64_t>()[i];
    problem_sizes[i] = trans_a
      ? cutlass::gemm::GemmCoord(hidden_in, hidden_out, bs)
      : cutlass::gemm::GemmCoord(bs, hidden_out, hidden_in);
  }
  return problem_sizes;
}

template <typename T>
torch::Tensor CopyToDevice(const std::vector<T> &x, const torch::Device &device) {
  size_t bytes = x.size() * sizeof(T);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(device);
  torch::Tensor out = torch::empty(bytes, options);

  CUDA_CALL(hipMemcpyAsync(out.data_ptr(),
			    x.data(), bytes,
			    hipMemcpyHostToDevice,
			    c10::cuda::getCurrentCUDAStream()));
  return out;
}

template <typename T>
static void ReorderArray(T* data, const std::vector<size_t>& indices) {
    // For now, simply create a copy of the data and then copy over to the original.
    std::vector<T> copy(data, data + indices.size());
    for (size_t i = 0; i < indices.size(); ++i) {
        data[i] = copy.at(indices[i]);
    }
}

template <typename Gemm, bool trans_a, bool trans_b>
typename Gemm::Arguments MakeArguments(torch::Tensor a,
				       torch::Tensor b,
				       torch::Tensor c,
				       torch::Tensor batch_sizes) {
  auto problem_sizes_host = MakeProblemSizes<trans_a, trans_b>(a, b, batch_sizes);

  int64_t num_experts_orig = problem_sizes_host.size();

  // Create the host arrays of leading dimension data and pointer data.
  using LayoutA = typename Gemm::LayoutA;
  using LayoutB = typename Gemm::LayoutB;
  using LayoutC = typename Gemm::LayoutC;

  std::vector<int64_t> lda_host, ldb_host, ldc_host;
  int64_t elements_a = 0, elements_b = 0, elements_c = 0;

  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementC = typename Gemm::ElementC;
  std::vector<ElementA *> ptr_a_host, ptr_b_host, ptr_c_host;

  lda_host.reserve(num_experts_orig);
  ldb_host.reserve(num_experts_orig);
  ldc_host.reserve(num_experts_orig);

  ptr_a_host.reserve(num_experts_orig);
  ptr_b_host.reserve(num_experts_orig);
  ptr_c_host.reserve(num_experts_orig);

  // CUTLASS doesn't handle problems with `k=0` correctly, see https://github.com/NVIDIA/cutlass/pull/1593.
  // Until a fix is available on the CUTLASS side, handle these problems by ourselves.
  int64_t num_experts = 0;
  for (int i = 0; i < num_experts_orig; ++i) {
    auto problem = problem_sizes_host[i];
    if (problem.k() == 0) {
      CUDA_CALL(hipMemsetAsync((ElementC*)c.data_ptr() + elements_c,
				0,
				problem.m() * problem.n() * sizeof(ElementC),
				c10::cuda::getCurrentCUDAStream()));
    } else {
      lda_host.push_back(LayoutA::packed({problem.m(), problem.k()}).stride(0));
      ldb_host.push_back(LayoutB::packed({problem.k(), problem.n()}).stride(0));
      ldc_host.push_back(LayoutC::packed({problem.m(), problem.n()}).stride(0));

      ptr_a_host.push_back((ElementA*)a.data_ptr() + elements_a);
      ptr_b_host.push_back((ElementB*)b.data_ptr() + elements_b);
      ptr_c_host.push_back((ElementC*)c.data_ptr() + elements_c);

      problem_sizes_host[num_experts++] = problem;
    }

    elements_a += problem.m() * problem.k();
    elements_b += problem.k() * problem.n();
    elements_c += problem.m() * problem.n();
  }
  problem_sizes_host.resize(num_experts);

  // Calculate the number of threadblocks to use and validate the result.
  // NOTE: This is borrowed from FasterTransformer.
  int threadblock_count = Gemm::sufficient(problem_sizes_host.data(), num_experts);
  if (!threadblock_count) {
    TORCH_CHECK(false, "Grouped GEMM execution not possible with HW");
  }

  // Only sort problems when trans_a = True because only this case K are different
  if (trans_a) {
      std::vector<size_t> indices(num_experts);
      std::iota(indices.begin(), indices.end(), 0);
      std::stable_sort(indices.begin(), indices.end(), [&problem_sizes_host](size_t i, size_t j) {
          return problem_sizes_host[i].k() > problem_sizes_host[j].k();
      });

      ReorderArray(problem_sizes_host.data(), indices);
      ReorderArray(lda_host.data(), indices);
      ReorderArray(ldb_host.data(), indices);
      ReorderArray(ldc_host.data(), indices);
      ReorderArray(ptr_a_host.data(), indices);
      ReorderArray(ptr_b_host.data(), indices);
      ReorderArray(ptr_c_host.data(), indices);
  }

  // Copy the problem sizes, pointers and leading dimension data to the device.
  torch::Tensor lda = CopyToDevice(lda_host, a.device());
  torch::Tensor ldb = CopyToDevice(ldb_host, a.device());
  torch::Tensor ldc = CopyToDevice(ldc_host, a.device());
  torch::Tensor ptr_a = CopyToDevice(ptr_a_host, a.device());
  torch::Tensor ptr_b = CopyToDevice(ptr_b_host, a.device());
  torch::Tensor ptr_c = CopyToDevice(ptr_c_host, a.device());
  torch::Tensor problem_sizes = CopyToDevice(problem_sizes_host, a.device());

  typename Gemm::EpilogueOutputOp::Params epilogue_op(/*alpha=*/1.0f, /*beta=*/0.0f);
  typename Gemm::Arguments arguments((cutlass::gemm::GemmCoord*)problem_sizes.data_ptr(),
  				     (int)num_experts,
  				     (int)threadblock_count,
  				     epilogue_op,
  				     (ElementA**)ptr_a.data_ptr(),
  				     (ElementB**)ptr_b.data_ptr(),
  				     (ElementC**)ptr_c.data_ptr(),
  				     (ElementC**)ptr_c.data_ptr(),
  				     /*lda=*/(int64_t*)lda.data_ptr(),
  				     /*ldb=*/(int64_t*)ldb.data_ptr(),
  				     /*ldc=*/(int64_t*)ldc.data_ptr(),
  				     /*ldd=*/(int64_t*)ldc.data_ptr(),
  				     (cutlass::gemm::GemmCoord*)problem_sizes_host.data());
  return arguments;
}

template <bool trans_a, bool trans_b>
torch::Tensor CutlassGroupedGemm(torch::Tensor a,
				 torch::Tensor b,
				 torch::Tensor c,
				 torch::Tensor batch_sizes) {
  using Gemm = GemmGrouped<trans_a, trans_b>;
  Gemm gemm;

  auto arguments = MakeArguments<Gemm, trans_a, trans_b>(a, b, c, batch_sizes);
  int64_t workspace_size = gemm.get_workspace_size(arguments);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(a.device());
  torch::Tensor workspace = torch::empty(workspace_size, options);

  // Initialize the kernel.
  if(gemm.initialize(arguments, workspace.data_ptr()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to initialize CUTLASS Grouped GEMM");
  }

  // Execute the kernel in the current stream.
  if(gemm.run(c10::cuda::getCurrentCUDAStream()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to run CUTLASS Grouped GEMM");
  }
  return c;
}

void CublasGemm(c10::BFloat16 *a, int64_t a_rows, int64_t a_cols, bool trans_a,
		c10::BFloat16 *b, int64_t b_rows, int64_t b_cols, bool trans_b,
		c10::BFloat16 *c, int64_t c_rows, int64_t c_cols) {
  int m = trans_b ? b_rows : b_cols;
  int k = trans_b ? b_cols : b_rows;
  int n = trans_a ? a_cols : a_rows;

  int lda = trans_a ? n : k;
  int ldb = trans_b ? k : m;
  hipblasOperation_t transpose_a = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transpose_b = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  float alpha = 1.0, beta = 0.0;
  CUBLAS_CALL(hipblasGemmEx(at::cuda::getCurrentCUDABlasHandle(),
			   transpose_b, transpose_a,
			   m, n, k, &alpha,
			   b, HIP_R_16BF, ldb,
			   a, HIP_R_16BF, lda,
			   &beta,
			   c, HIP_R_16BF, c_cols, HIP_R_32F,
			   HIPBLAS_GEMM_DEFAULT));
}

void CublasGroupedGemm(torch::Tensor a,
		       torch::Tensor b,
		       torch::Tensor c,
		       torch::Tensor batch_sizes,
		       bool trans_b) {
  int64_t bs = batch_sizes.size(0), k = a.size(1);
  int64_t n = trans_b ? b.size(1) : b.size(2);
  int64_t b_rows = b.size(1), b_cols = b.size(2);
  c10::BFloat16* a_ptr = a.data_ptr<c10::BFloat16>();
  c10::BFloat16* b_ptr = b.data_ptr<c10::BFloat16>();
  c10::BFloat16* c_ptr = c.data_ptr<c10::BFloat16>();
  for (int i = 0; i < bs; ++i) {
    int64_t m = batch_sizes.data_ptr<int64_t>()[i];
    CublasGemm(a_ptr, m, k, /*trans_a=*/false,
	       b_ptr, b_rows, b_cols, trans_b,
	       c_ptr, m, n);
    a_ptr += m * k;
    b_ptr += b_rows * b_cols;
    c_ptr += m * n;
  }
}

void CublasGroupedGemmVariableK(torch::Tensor a,
				torch::Tensor b,
				torch::Tensor c,
				torch::Tensor batch_sizes) {
  int64_t bs = batch_sizes.size(0), m = a.size(1), n = b.size(1);
  c10::BFloat16* a_ptr = a.data_ptr<c10::BFloat16>();
  c10::BFloat16* b_ptr = b.data_ptr<c10::BFloat16>();
  c10::BFloat16* c_ptr = c.data_ptr<c10::BFloat16>();
  for (int i = 0; i < bs; ++i) {
    int64_t k = batch_sizes.data_ptr<int64_t>()[i];
    CublasGemm(a_ptr, k, m, /*trans_a=*/true,
	       b_ptr, k, n, /*trans_b=*/false,
	       c_ptr, m, n);
    a_ptr += k * m;
    b_ptr += k * n;
    c_ptr += m * n;
  }
}

void GroupedGemmVariableK(torch::Tensor a,
			  torch::Tensor b,
			  torch::Tensor c,
			  torch::Tensor batch_sizes) {
  // We expected a CUDA tensor with two dimensions and shape
  // (tokens, hidden_out) for 'b'.
  TORCH_CHECK(b.is_cuda());
  TORCH_CHECK(b.ndimension() == 2);
  TORCH_CHECK(b.scalar_type() == torch::kBFloat16);

  // Validate the dimensions.
  int64_t tokens = a.size(0), num_experts = batch_sizes.size(0);
  int64_t m = a.size(1), n = b.size(1);

  // Validate that we have the same contraction dimension.
  TORCH_CHECK(tokens == b.size(0));

  // Validate the output shape.
  TORCH_CHECK(c.is_cuda());
  TORCH_CHECK(c.ndimension() == 3);
  TORCH_CHECK(c.scalar_type() == torch::kBFloat16);
  TORCH_CHECK(c.size(0) == num_experts);
  TORCH_CHECK(c.size(1) == m);
  TORCH_CHECK(c.size(2) == n);

  // Run the computation.
  CublasGroupedGemmVariableK(a, b, c, batch_sizes);
}

// NOTE: We only support dynamic group sizes for the 'a' tensor. Tensor 'b' is
// assumed to be batched with fixed sized batches.
//
// TODO(tgale): Validate alignment is true for every batch element.
void GroupedGemm(torch::Tensor a,
		 torch::Tensor b,
		 torch::Tensor c,
		 torch::Tensor batch_sizes,
		 bool trans_a, bool trans_b) {
  // NOTE: We only support 'trans_a' or 'trans_b', not both.
  TORCH_CHECK(!(trans_a && trans_b));

  // We expect the batch_sizes on CPU.
  TORCH_CHECK(batch_sizes.is_cpu());
  TORCH_CHECK(batch_sizes.ndimension() == 1);
  TORCH_CHECK(batch_sizes.scalar_type() == torch::kInt64);

  // We expected a CUDA tensor with two dimensions and shape
  // (tokens, hidden_in) for 'a'.
  TORCH_CHECK(a.is_cuda());
  TORCH_CHECK(a.ndimension() == 2);
  TORCH_CHECK(a.scalar_type() == torch::kBFloat16);

#if !defined(GROUPED_GEMM_CUTLASS)
  if (trans_a) {
    // If we can't use CUTLASS for the transposed cases, defer to the variable 'k' helper using cuBLAS
    // for the rest of the op.
    GroupedGemmVariableK(a, b, c, batch_sizes);
    return;
  }
#endif

  TORCH_CHECK(b.is_cuda());
  TORCH_CHECK(c.is_cuda());
  TORCH_CHECK(b.scalar_type() == torch::kBFloat16);
  TORCH_CHECK(c.scalar_type() == torch::kBFloat16);

  // The expected shapes of 'b' and 'c' are:
  //   * when 'trans_a' is set: b=(tokens, hidden_out),                 c=(num_experts, hidden_in, hidden_out)
  //   * when 'trans_b' is set: b=(num_experts, hidden_out, hidden_in), c=(tokens, hidden_out)
  //   * otherwise:             b=(num_experts, hidden_in, hidden_out), c=(tokens, hidden
  if (trans_a) {
    TORCH_CHECK(b.ndimension() == 2);
    TORCH_CHECK(c.ndimension() == 3);
    TORCH_CHECK(b.size(0) == a.size(0));
    TORCH_CHECK(c.size(0) == batch_sizes.size(0));
    TORCH_CHECK(c.size(1) == a.size(1));
    TORCH_CHECK(c.size(2) == b.size(1));
  } else {
    TORCH_CHECK(b.ndimension() == 3);
    TORCH_CHECK(c.ndimension() == 2);

    // Validate the contraction dimensions match.
    int64_t tokens = a.size(0), num_experts = b.size(0);
    int64_t hidden_in = trans_b ? b.size(2) : b.size(1);
    int64_t hidden_out = trans_b ? b.size(1) : b.size(2);
    TORCH_CHECK(hidden_in == a.size(1));

    // Validate that we have one size per expert.
    TORCH_CHECK(batch_sizes.size(0) == num_experts);
  }

  // NOTE: We support transposition through the 'trans_b' flag.
  TORCH_CHECK(a.is_contiguous());
  TORCH_CHECK(b.is_contiguous());
  TORCH_CHECK(c.is_contiguous());

#if !defined(GROUPED_GEMM_CUTLASS)
  CublasGroupedGemm(a, b, c, batch_sizes, trans_b);
  return;
#else
  if (trans_a) {
    CutlassGroupedGemm<true, false>(a, b, c, batch_sizes);
    return;
  }
  if (trans_b) {
    CutlassGroupedGemm<false, true>(a, b, c, batch_sizes);
    return;
  }
  CutlassGroupedGemm<false, false>(a, b, c, batch_sizes);
  return;
#endif
}

}  // namespace grouped_gemm
