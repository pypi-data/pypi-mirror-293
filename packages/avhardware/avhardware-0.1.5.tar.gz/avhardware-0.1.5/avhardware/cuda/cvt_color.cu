#include <stdint.h>
#include <hip/hip_runtime.h>


__global__ void cuda_nv12_to_rgb(uint8_t *in_y, uint8_t *in_uv, uint8_t *out_rgb, int height, int width, int pitch, int full_color_range) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int y_index = y * pitch + x;
    int uv_index = (y / 2) * pitch + (x / 2) * 2;

    uint8_t Y = in_y[y_index];
    uint8_t U = in_uv[uv_index];
    uint8_t V = in_uv[uv_index + 1];

    int C = Y - 16;
    int D = U - 128;
    int E = V - 128;

    uint8_t R = min(max((298 * C + 409 * E + 128) >> 8, 0), 255);
    uint8_t G = min(max((298 * C - 100 * D - 208 * E + 128) >> 8, 0), 255);
    uint8_t B = min(max((298 * C + 516 * D + 128) >> 8, 0), 255);

    int rgb_index = (y * width + x) * 3;
    out_rgb[rgb_index] = R;
    out_rgb[rgb_index + 1] = G;
    out_rgb[rgb_index + 2] = B;
}


// Host function to launch the CUDA kernel
extern "C" {
    hipError_t nv12_to_rgb(uint8_t *in_y, uint8_t *in_uv, uint8_t *out_rgb, int height, int width, int pitch, int full_color_range) {
        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        cuda_nv12_to_rgb<<<grid, block>>>(in_y, in_uv, out_rgb, height, width, pitch, full_color_range);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) return err;
        err = hipDeviceSynchronize();
        if (err != hipSuccess) return err;
        return hipSuccess;
    }
}
