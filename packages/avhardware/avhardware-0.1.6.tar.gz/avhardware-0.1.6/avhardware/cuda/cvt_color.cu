#include <stdint.h>
#include <hip/hip_runtime.h>


template<class T>
__device__ static T clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<bool FullColorRange>
__global__ void cudaNV12ToRGB(
    uint8_t *in_y,
    uint8_t *in_uv,
    uint8_t *out_rgb,
    int height,
    int width,
    int pitch
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int y_index = y * pitch + x;
    int uv_index = (y / 2) * pitch + (x / 2) * 2;

    uint8_t Y = in_y[y_index];
    uint8_t U = in_uv[uv_index];
    uint8_t V = in_uv[uv_index + 1];

    float fY = (int)Y - 0;
    float fU = (int)U - 128;
    float fV = (int)V - 128;

    uint8_t R, G, B;
    if constexpr (FullColorRange) {
        R = clamp(1.000f * fY +             + 1.402f * fV, 0.0f, 255.0f);
        G = clamp(1.000f * fY - 0.344f * fU - 0.714f * fV, 0.0f, 255.0f);
        B = clamp(1.000f * fY + 1.772f * fU              , 0.0f, 255.0f);
    } else {
        fY -= 16;
        R = clamp(1.164f * fY +             + 1.596f * fV, 0.0f, 255.0f);
        G = clamp(1.164f * fY - 0.392f * fU - 0.813f * fV, 0.0f, 255.0f);
        B = clamp(1.164f * fY + 2.017f * fU              , 0.0f, 255.0f);
    }

    int rgb_index = (y * width + x) * 3;
    out_rgb[rgb_index] = R;
    out_rgb[rgb_index + 1] = G;
    out_rgb[rgb_index + 2] = B;
}


// Host function to launch the CUDA kernel
extern "C" {
    hipError_t NV12ToRGB(uint8_t *in_y, uint8_t *in_uv, uint8_t *out_rgb, int height, int width, int pitch, bool full_color_range) {
        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        if (full_color_range) {
            cudaNV12ToRGB<true><<<grid, block>>>(in_y, in_uv, out_rgb, height, width, pitch);
        } else {
            cudaNV12ToRGB<false><<<grid, block>>>(in_y, in_uv, out_rgb, height, width, pitch);
        }

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) return err;
        err = hipDeviceSynchronize();
        if (err != hipSuccess) return err;
        return hipSuccess;
    }
}
